#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <sys/time.h>

#include "radixsort.h"
#include "random.hpp"
#include "timer.h"

#include "bsearch_cuda.h"

int main(int argc, char *argv[]) {


	if (argc < 4) {
			fprintf(stderr, "usage: %s <D size> <Q size> <I size>"
					"<seed> <device>\n",
					argv[0]);
			return 1;
	}

	CUDA_SAFE_CALL( hipSetDevice( atoi(argv[5] ) ) );
	//CUDA_SAFE_CALL( hipFree(NULL) );

	int D_size = atoi(argv[1]);
	int Q_size = atoi(argv[2]);
	int I_size = atoi(argv[3]);
	int seed = atoi(argv[4]);
	hipError_t err;

	//{{{ gen Q and D
	RNG_rand48 D_r(seed);
	D_r.generate(D_size);
	unsigned int *D_d = (unsigned int *)D_r.get_random_numbers();

	RNG_rand48 Q_r(seed);
	Q_r.generate(Q_size);
	unsigned int *Q_d = (unsigned int *)Q_r.get_random_numbers();
	
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "rand errors: %s.\n", hipGetErrorString( err) );
	//}}}

	//{{{ sort D
	start();
	nvRadixSort::RadixSort sort_D_d(D_size, true);
	sort_D_d.sort((unsigned int*)D_d, 0, D_size, 32);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "sort d: %s.\n", hipGetErrorString( err) );

	stop();
	unsigned long sort_d_time = report();
	//}}}

	/*
	unsigned int *D_h = (unsigned int *)malloc( D_size * sizeof(unsigned int));
	hipMemcpy(D_h, D_d, (D_size) * sizeof(unsigned int),
			hipMemcpyDeviceToHost);
	*/

	int block_size = 256;
	dim3 dimBlock(block_size);

	int index_grid_size = ( I_size + block_size - 1) / (block_size * 1);
	dim3 index_dimGrid( index_grid_size );

	unsigned int *I_d;
	hipMalloc((void **)&I_d, (I_size)*sizeof(unsigned int));

	//{{{ index
	start();
	gen_index <<<index_dimGrid, dimBlock>>> ( D_d, D_size, I_d, I_size);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "index: %s.\n", hipGetErrorString( err) );
	stop();
	unsigned long index_time = report();
	//}}}

	/*
	unsigned int *I_h = (unsigned int *) malloc(I_size*sizeof(unsigned int));
	hipMemcpy(I_h, I_d, (I_size) * sizeof(unsigned int),
			hipMemcpyDeviceToHost);
	*/
	

	int grid_size = ( Q_size + block_size - 1) / (block_size * 1);
	dim3 dimGrid( grid_size );

	unsigned int *R_d;
	hipMalloc((void **)&R_d, (Q_size)*sizeof(unsigned int));

	//{{{ i_gm_binary_search
	start();
	i_gm_binary_search<<< dimGrid, dimBlock>>> (
			D_d, D_size, Q_d, Q_size, R_d, I_d, I_size);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "binary_search_gp 1: %s.\n", hipGetErrorString( err) );

	stop();
	unsigned long search_gmindex_1_time = report();

	start();
	i_gm_binary_search<<< dimGrid, dimBlock>>> (
			D_d, D_size, Q_d, Q_size, R_d, I_d, I_size);

	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess)
		fprintf(stderr, "binary_search_gp 2: %s.\n", hipGetErrorString( err) );

	stop();
	unsigned long search_gmindex_2_time = report();
	//}}}

	printf("%lu,%lu\n", 
			search_gmindex_1_time + index_time,
			search_gmindex_2_time + index_time);

	return 0;
}

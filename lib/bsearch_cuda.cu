#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "bsearch_cuda.h"

//{{{ __global__ void b_search( unsigned int *db,
__global__
void b_search( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R )
				     
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if ( id < size_q )
		R[id] = bound_binary_search(db, size_db, q[id], -1, size_db );
}
//}}}

//{{{ __global__ void gen_index( unsigned int *db,
__global__
void gen_index( unsigned int *db,
			    int size_db, 
				unsigned int *I,
				int size_I)
				     
{
	//extern __shared__ unsigned int I[];

	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i;
	if ( id < size_I) {
		i = i_to_I(id, size_I, size_db);
		I[id] = db[i];
	}
}
//}}}

//{{{ __global__ void gen_tree( unsigned int *db,
__global__
void gen_tree( unsigned int *db,
			    int size_db, 
				unsigned int *T,
				int size_T)
				     
{
	//extern __shared__ unsigned int I[];

	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i;
	if ( id < size_T) {
		i = i_to_T(id, size_T, size_db);
		T[id] = db[i];
	}
}
//}}}

//{{{int i_to_T(int i, int T_size, int D_size)
__device__
unsigned int i_to_T(int i, int T_size, int D_size)
{
	int hi = D_size;
	double row_d = logf(i + 1) / logf(2);
	unsigned long int row = (int) (row_d);
	unsigned long int prev = powf(2, row) - 1;
	unsigned long int i_row = i - prev;

	unsigned long int hi_v = 2*i_row + 1;
	unsigned long int lo_v = powf(2, row + 1) - (2*i_row +1);
	unsigned long int div = powf(2,row + 1);
	unsigned long int r = ( hi_v*hi - lo_v) / div;

	//printf("hi:%d\tlo:%d\trow:%u\thi_v:%u\tlo_v:%u\tdiv:%u\tr:%u\n",
			//hi, lo, row, hi_v, lo_v, div, r);
	return r; 
}       
//}}}       

//{{{int i_to_I(int i, int I_size, int D_size)
__device__
unsigned int i_to_I(int i, int I_size, int D_size)
{
	unsigned long int regions = I_size + 1;
	unsigned long int hi = D_size;
	unsigned long int j = i;
	unsigned int r =( (j+1)*hi - (regions - (j+1))) / (regions);
	return r;
}
//}}}

//{{{ __global__ void i_sm_binary_search( unsigned int *db,
__global__
void i_sm_binary_search( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R,
					 unsigned int *I,
					 int size_I)
				     
{
	extern __shared__ unsigned int L[];

	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int c, round = 0;

	while ( ( (blockDim.x * round) + threadIdx.x ) < size_I) {
		c = (blockDim.x*round) + threadIdx.x;
		L[c] = I[c];
		++round;
	}
	__syncthreads();

	if (id < size_q) {
		int key = q[id];
		int b = binary_search_cuda(L, size_I, key);

		int new_hi, new_lo;
		region_to_hi_lo(b, size_I + 1, size_db, &new_hi, &new_lo);
		unsigned int x =  bound_binary_search(
				db, size_db, key, new_lo, new_hi);
		R[id] = x;



		/*
		int new_hi = ( (b+1)*size_db - (size_I - (b+2))) / size_I;
		int new_lo = ( (b  )*size_db - (size_I - (b+1))) / size_I;

		if (b == 0)
			new_lo = -1;
		else if (b == size_I) {
			new_hi = size_db;
			//new_lo = ( (b-1)*size_db + (I_size - (b+1))*lo ) / I_size;
			new_lo = ( (b-1)*size_db - (size_I - (b)) ) / size_I;
		}

		R[id] =  bound_binary_search(db, size_db, key, new_lo, new_hi);
		*/
	}
}
//}}}

//{{{ __global__ void i_gm_binary_search( unsigned int *db,
__global__
void i_gm_binary_search( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R,
					 unsigned int *I,
					 int size_I)
				     
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (id < size_q) {
		unsigned long int key = q[id];
		unsigned long int b = bound_binary_search(I, size_I, key, -1, size_I);
		int new_hi, new_lo;
		region_to_hi_lo(b, size_I + 1, size_db, &new_hi, &new_lo);
		unsigned int x =  bound_binary_search(
				db, size_db, key, new_lo, new_hi);
		R[id] = x;
	}
}
//}}}

//{{{ __device__ int i_binary_search( unsigned int *db,
__device__
int i_binary_search( unsigned int *db,
					 int size_db, 
					 unsigned int s,
					 unsigned int *I,
					 int size_I)
				     
{
	unsigned int b = bound_binary_search(I, size_I, s, -1, size_I);
	int new_hi, new_lo;
	region_to_hi_lo(b, size_I + 1, size_db, &new_hi, &new_lo);
	return bound_binary_search(db, size_db, s, new_lo, new_hi);
}
//}}}


//{{{ __global__ void t_sm_binary_search( unsigned int *db,
__global__
void t_sm_binary_search( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R,
					 unsigned int *T,
					 int size_T)
				     
{
	extern __shared__ unsigned int L[];

	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int c, round = 0;

	while ( ( (blockDim.x * round) + threadIdx.x ) < size_T) {
		c = (blockDim.x*round) + threadIdx.x;
		L[c] = T[c];
		++round;
	}
	__syncthreads();


	if (id < size_q) {
		int key = q[id];

		unsigned long int b = 0;
		unsigned long int t = 0;

		while (b < size_T) {
			t = L[b];
			if (key < t)
				b = 2*(b) + 1;
			else if (key > t)
				b = 2*(b) + 2;
			else
				break;
		}

		if (t == key)
			R[id] = i_to_T(b, size_T, size_db);
		else {
			int new_hi, new_lo;
			region_to_hi_lo(b - size_T, size_T + 1, size_db, &new_hi, &new_lo);
			unsigned int x =  bound_binary_search(
					db, size_db, key, new_lo, new_hi);
			R[id] = x;
		}
	}
}
//}}}

//{{{ __global__ void t_gm_binary_search( unsigned int *db,
__global__
void t_gm_binary_search( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R,
					 unsigned int *T,
					 int size_T)
				     
{
	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < size_q) {
		int key = q[id];

		unsigned long int b = 0;
		unsigned long int t = 0;

		while (b < size_T) {
			t = T[b];
			if (key < t)
				b = 2*(b) + 1;
			else if (key > t)
				b = 2*(b) + 2;
			else
				break;
		}

		/*
			R[id] = b - size_T;
		*/
		if (t == key)
			R[id] = i_to_T(b, size_T, size_db);
		else {
			int new_hi, new_lo;
			region_to_hi_lo(b - size_T, size_T + 1, size_db, &new_hi, &new_lo);
			unsigned int x =  bound_binary_search(
					db, size_db, key, new_lo, new_hi);
			R[id] = x;
		}
	}
}
//}}}

/*
//{{{ __global__ void binary_search_i( unsigned int *db,
__global__
void binary_search_i( unsigned int *db,
					 int size_db, 
					 unsigned int *q,
					 int size_q, 
					 unsigned int *R,
					 int size_I)
				     
{
	extern __shared__ unsigned int I[];

	unsigned int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	int i, c, round = 0;

	while ( ( (blockDim.x * round) + threadIdx.x ) < size_I) {
		c = (blockDim.x*round) + threadIdx.x;
		i = ((c + 1)*size_db - (size_I - c + 1))/size_I;
		I[c] = db[i];
		++round;
		//if ( blockIdx.x == 0 )
			//R[c] = I[c];
	}
	__syncthreads();

	if (id < size_q) {
		int key = q[id];
		int b = binary_search_cuda(I, size_I, key);

		int new_hi = ( (b+1)*size_db - (size_I - (b+2))) / size_I;
		int new_lo = ( (b  )*size_db - (size_I - (b+1))) / size_I;

		if (b == 0)
			new_lo = -1;
		else if (b == size_I) {
			new_hi = size_db;
			//new_lo = ( (b-1)*size_db + (I_size - (b+1))*lo ) / I_size;
			new_lo = ( (b-1)*size_db - (size_I - (b)) ) / size_I;
		}

		R[id] =  bound_binary_search(db, size_db, key, new_lo, new_hi);
		//R[id] =  id;
	}
}
//}}}
*/

//{{{ __device__ int bound_binary_search( unsigned int *db,
__device__
int bound_binary_search( unsigned int *db,
				   int size_db, 
				   unsigned int s,
				   int lo,
				   int hi) 
{
	int mid;
	while ( hi - lo > 1) {
		mid = (hi + lo) / 2;

		if ( db[mid] < s )
			lo = mid;
		else
			hi = mid;
	}
	return hi;
}
//}}}

//{{{ __device__ int binary_search_cuda( unsigned int *db, int size_db, unsigned int
__device__
int binary_search_cuda( unsigned int *db,
				   int size_db, 
				   unsigned int s) 
{
	int lo = -1, hi = size_db, mid;
	while ( hi - lo > 1) {
		mid = (hi + lo) / 2;

		if ( db[mid] < s )
			lo = mid;
		else
			hi = mid;
	}
	return hi;
}
//}}}

//{{{ __device__ void region_to_hi_lo(int region, int I_size, int D_size, int
__device__
void region_to_hi_lo(int region, int I_size, int D_size, int *D_hi, int *D_lo)
{
	unsigned long int hi = D_size;
	unsigned long int r = region;

	unsigned long int l_new_hi = ((r+1)*hi - (I_size - (r+1)) ) / I_size;
	//int new_hi = ( (region+1)*hi - (I_size - (region+1)) ) / I_size;
	//int new_lo = (( (region)*hi - (I_size - (region+1)) ) / I_size) - 1;
	unsigned long int l_new_lo = (((r)*hi - (I_size - (r+1)) ) / I_size) - 1;

	int new_hi = l_new_hi, new_lo = l_new_lo;

	if (region == 0) {
		new_hi = l_new_hi;
		new_lo = -1;
	} else if (region == I_size) {
		new_hi = D_size;
		l_new_lo = ( (r-1)*hi - (I_size - (r+1)) ) / I_size;
		new_lo = l_new_lo;
	}
	
	*D_hi = new_hi;
	*D_lo = new_lo;
}
//}}}

//{{{int i_to_T(int i, int T_size, int D_size)
unsigned int _i_to_T(int i, int T_size, int D_size)
{
	int hi = D_size;
	double row_d = logf(i + 1) / logf(2);
	unsigned int row = (int) (row_d);
	unsigned int prev = powf(2, row) - 1;
	unsigned int i_row = i - prev;

	unsigned int hi_v = 2*i_row + 1;
	unsigned int lo_v = powf(2, row + 1) - (2*i_row +1);
	unsigned int div = powf(2,row + 1);
	unsigned int r = ( hi_v*hi - lo_v) / div;

	//printf("hi:%d\tlo:%d\trow:%u\thi_v:%u\tlo_v:%u\tdiv:%u\tr:%u\n",
			//hi, lo, row, hi_v, lo_v, div, r);
	return r; 
}       
//}}}       

//{{{int i_to_I(int i, int I_size, int D_size)
unsigned int _i_to_I(int i, int I_size, int D_size)
{
	unsigned long int regions = I_size + 1;
	unsigned long int hi = D_size;
	unsigned long int j = i;
	unsigned int r =( (j+1)*hi - (regions - (j+1))) / (regions);

	/*
	unsigned long int a =(j+1)*hi;
	unsigned long int b = regions - (j+1);

	printf("\t%lu\t%lu\t%lu\n", a, b, regions);
	*/
	return r;
}
//}}}


